#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <cmath>
#include "visimerge/kernel_visimergesort.cuh"
#include "visimerge/io_common.h"

using namespace vmgpu;

template<typename T>
std::vector<viewray<T>> solve_visibility_gpu(const std::vector<segment<T>> &host_segments, bool profile = false)
{
    mgpu::standard_context_t context(false);

    const int seg_count = host_segments.size();
    const int vr_count = 2 * seg_count;

    mgpu::mem_t<segment<T>> dev_segments(seg_count, context);
    vray_array<T> dev_vrays = vray_array<T>::create(vr_count, context);

    mgpu::htod(dev_segments.data(), host_segments.data(), seg_count);

    kernel_visimergesort(dev_segments, dev_vrays, context, profile);
    context.synchronize();

    vray_array<T> host_vrays = vray_array<T>::create(vr_count, context, mgpu::memory_space_host);

    mgpu::dtoh(host_vrays.t, dev_vrays.t, vr_count);
    mgpu::dtoh(host_vrays.l, dev_vrays.l, vr_count);
    mgpu::dtoh(host_vrays.r, dev_vrays.r, vr_count);

    std::vector<viewray<T>> vrays_vec(vr_count);

    for (int i = 0; i < vr_count; ++i)
        vrays_vec[i] = { host_vrays.t[i], host_vrays.l[i], host_vrays.r[i] };

    vray_array<T>::destroy(dev_vrays, context);
    vray_array<T>::destroy(host_vrays, context, mgpu::memory_space_host);

    return vrays_vec;
}


int main(int argc, char** argv)
{
    typedef float real_t;

    if (argc < 2)
    {
        std::cerr << argv[0] << ": missing file operand" << std::endl;
        return EXIT_FAILURE;
    }

    auto segments = readfile<real_t>(argv[1]);

    if (1u << mgpu::find_log2(segments.size(), true) != segments.size())
    {
        std::cerr << "current visimergesort only works for segment sets with 2^k segments" << std::endl;
        return EXIT_FAILURE;
    }

    bool profile = argc >= 3 && std::string(argv[2]) == "--profile";

    auto vrays_vec = solve_visibility_gpu(segments, profile);

    if (!profile) print_viewrays(vrays_vec, std::cout);

    return 0;
}
