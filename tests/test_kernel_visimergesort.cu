#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <cmath>
#include <sys/time.h>
#include <unistd.h>
#include "visimerge/kernel_visimergesort.cuh"

using namespace vmgpu;


template<typename T>
std::vector<segment<T>> readfile(const std::string &filename)
{
    typedef segment<T> segmentT;

    std::ifstream file(filename.c_str());
    std::string line;

    std::vector<segmentT> vec;

    while (std::getline(file, line))
    {
        std::stringstream ss(line);

        segmentT seg;

        ss >> seg.a.x;
        ss.get();
        ss >> seg.a.y;

        ss >> seg.b.x;
        ss.get();
        ss >> seg.b.y;

        vec.push_back(seg);
    }

    return vec;
}


template<typename T, typename Os>
void print_viewrays(const std::vector<viewray<T>> &vec, Os &out)
{
    for (int i = 0; i < vec.size(); ++i)
    {
        out << vec[i].t << " "
            << (almost_equal(vec[i].vx(), 0.0) ? 0.0 : vec[i].vx()) << " "
            << (almost_equal(vec[i].vy(), 0.0) ? 0.0 : vec[i].vy()) << " "
            << (std::isinf(vec[i].r) ? -1 : vec[i].r) << " "
            << (std::isinf(vec[i].l) ? -1 : vec[i].l) << std::endl;
    }
}


int main(int argc, char** argv)
{
    if (argc < 2)
    {
        std::cerr << argv[0] << ": missing file operand" << std::endl;
        return EXIT_FAILURE;
    }

    auto vec = readfile<double>(argv[1]);

    if (1u << mgpu::find_log2(vec.size(), true) != vec.size())
    {
        std::cerr << "current visimergesort only works for segment sets with 2^k segments" << std::endl;
        return EXIT_FAILURE;
    }

    bool profile = argc >= 3 && std::string(argv[2]) == "--profile";

    std::vector<viewray<double>> vis(vec.size() * 2);

    mgpu::standard_context_t context(false);

    struct timeval start, end;

    if (profile) gettimeofday(&start, NULL);

    kernel_visimergesort(vec.data(), vec.size(), vis.data(), context, profile);

    if (profile)
    {
        gettimeofday(&end, NULL);

        long seconds  = end.tv_sec  - start.tv_sec;
        long useconds = end.tv_usec - start.tv_usec;

        double s = seconds + useconds / 1e6;

        std::cerr << "kernel_visimergesort took " << s * 1e3 << "ms to find the visibility region of " << vec.size()
                  << " segments" << std::endl;
    }

    print_viewrays(vis, std::cout);

    return 0;
}
