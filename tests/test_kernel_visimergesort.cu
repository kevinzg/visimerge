#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <cmath>
#include "visimerge/kernel_visimergesort.cuh"

using namespace vmgpu;


template<typename T>
std::vector<segment<T>> readfile(const std::string &filename)
{
    typedef segment<T> segmentT;

    std::ifstream file(filename.c_str());
    std::string line;

    std::vector<segmentT> vec;

    while (std::getline(file, line))
    {
        std::stringstream ss(line);

        segmentT seg;

        ss >> seg.a.x;
        ss.get();
        ss >> seg.a.y;

        ss >> seg.b.x;
        ss.get();
        ss >> seg.b.y;

        vec.push_back(seg);
    }

    return vec;
}


template<typename T, typename Os>
void print_viewrays(const std::vector<viewray<T>> &vec, Os &out)
{
    for (int i = 0; i < vec.size(); ++i)
    {
        out << vec[i].t << " "
            << vec[i].v.x << " "
            << vec[i].v.y << " "
            << (std::isinf(vec[i].r) ? -1 : vec[i].r) << " "
            << (std::isinf(vec[i].l) ? -1 : vec[i].l) << std::endl;
    }
}


int main(int argc, char** argv)
{
    if (argc < 2)
    {
        std::cerr << argv[0] << ": missing file operand" << std::endl;
        return EXIT_FAILURE;
    }

    auto vec = readfile<double>(argv[1]);

    if (1u << mgpu::find_log2(vec.size(), true) != vec.size())
    {
        std::cerr << "current visimergesort only works for segment sets with 2^k segments" << std::endl;
        return EXIT_FAILURE;
    }

    bool profile = argc >= 3 && std::string(argv[2]) == "--profile";

    std::vector<viewray<double>> vis(vec.size() * 2);

    mgpu::standard_context_t context(false);

    kernel_visimergesort(vec.data(), vec.size(), vis.data(), context, profile);

    print_viewrays(vis, std::cout);

    return 0;
}
